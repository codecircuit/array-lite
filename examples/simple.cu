#include "hip/hip_runtime.h"
#include <nonstd/array.h>
#include <array>

#include <iostream>
#include <cstdio>

__global__ void kernel(nonstd::array<float, 3> arr) {
	for (auto& e : arr) {
		printf("e = %f\n", e);
	}

	nonstd::array<float, 3> other;
	other = nonstd::array<float, 3>({10, 11, 12});
	printf("other = %f %f %f\n", other[0], other[1], other[2]);
}

int main() {


	nonstd::array<int, 2> defconstr;

	nonstd::array<float, 3> arr = {1, 2, 3};
	nonstd::array<float, 3> abc({8, 9, 10});
	std::array<float, 3> stdarr = {8, 9, 10};
	kernel<<<1,1>>>(arr);
	hipDeviceSynchronize();

	// compatible with std::array
	arr = stdarr;
}
